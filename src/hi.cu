#include "hip/hip_runtime.h"
#include <fmt/core.h>
#include <iterator>
#include "io.hpp"
#include "device_timer.cuh"
#include "util.hpp"
#include <cxxopts.hpp>
#include <thrust/scan.h>
#include "helpers.cuh"

// Adapted from https://github.com/concept-inversion/H-INDEX_Triangle_Counting
__device__
int linearSearch(unsigned int neighbor, const unsigned int* partition1, const unsigned int* binCounts, unsigned int bin,
        unsigned int BIN_OFFSET, unsigned int BIN_START, unsigned int BUCKETS)
{
    unsigned int len = binCounts[bin + BIN_OFFSET];
    unsigned int i = bin + BIN_START;
    unsigned int step = 0;
    while(step < len)
    {
        unsigned int test = partition1[i];
        if(test==neighbor)
        {
            return 1;
        }
        else
        {
            i+=BUCKETS;
        }
        step += 1;
    }
    return 0;
}

__global__ void warpBasedHI(unsigned int numOfSets, const unsigned int* elements, const unsigned int* sizes,
        const unsigned int* offsets, unsigned int* counts, unsigned int* bins, unsigned int numOfBuckets, unsigned int bucketSize);

template <bool split>
__global__ void blockBasedHI(unsigned int numOfSets, const unsigned int* elements, const unsigned int* sizes,
        const unsigned int* offsets, unsigned int* counts, unsigned int* bins, unsigned int numOfBuckets, unsigned int bucketSize);

__global__ void intersectPathHI(unsigned int numOfSets, const unsigned int* elements, const unsigned int* offsets,
        unsigned int* counts, unsigned int* bins, unsigned int numOfBuckets,
        unsigned int bucketSize, unsigned int* globalDiagonals);

int main(int argc, char** argv) {
    try {
        fmt::print("{}\n", "Hash-based GPU set intersection");

        int multiprocessorCount;
        int maxThreadsPerBlock;

        hipDeviceGetAttribute(&multiprocessorCount, hipDeviceAttributeMultiprocessorCount, 0);
        hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);

        // arguments
        std::string input;
        std::string output;
        unsigned int blocks = multiprocessorCount * 16;
        unsigned int blockSize = maxThreadsPerBlock / 2;
        unsigned int buckets = 512;
        unsigned int bucketSize = 1000;
        bool warpBased = false;
        bool partition = false;

        cxxopts::Options options(argv[0], "Help");

        options.add_options()
                ("input", "Input dataset path", cxxopts::value<std::string>(input))
                ("output", "Output result path", cxxopts::value<std::string>(output))
                ("blocks", "Number of blocks (default: " + std::to_string(blocks) + ")", cxxopts::value<unsigned int>(blocks))
                ("threads", "Threads per block (default: " + std::to_string(blockSize) + ")", cxxopts::value<unsigned int>(blockSize))
                ("buckets", "Number of buckets (default: 512)", cxxopts::value<unsigned int>(buckets))
                ("bucket-size", "Size of each bucket (default: 1000)", cxxopts::value<unsigned int>(bucketSize))
                ("warp", "Launch warp based kernel (default: false, runs block based)", cxxopts::value<bool>(warpBased))
                ("path", "Adapt intersect path 1st level partitioning to distribute workload across thread blocks", cxxopts::value<bool>(partition))
                ("help", "Print help");

        auto result = options.parse(argc, argv);

        if (result.count("help")) {
            fmt::print("{}\n", options.help());
            return 0;
        }

        if (!result.count("input")) {
            fmt::print("{}\n", "No input dataset given! Exiting...");
            return 1;
        }

        Dataset* d = readDataset(input);

        fmt::print(
                "┌{0:─^{1}}┐\n"
                "│{3: ^{2}}|{4: ^{2}}│\n"
                "│{5: ^{2}}|{6: ^{2}}│\n"
                "│{7: ^{2}}|{8: ^{2}}│\n"
                "└{9:─^{1}}┘\n", "Dataset characteristics", 51, 25,
                "Cardinality", d->cardinality,
                "Universe", d->universe,
                "Total elements", d->totalElements, ""
        );


        unsigned int warpsPerBlock = blockSize / 32; // 32 is the warp size
        unsigned int binsMemory = blocks * warpsPerBlock * buckets * bucketSize * sizeof(unsigned int);

        fmt::print(
                "┌{0:─^{1}}┐\n"
                "│{3: ^{2}}|{4: ^{2}}│\n"
                "│{5: ^{2}}|{6: ^{2}}│\n"
                "│{7: ^{2}}|{8: ^{2}}│\n"
                "│{9: ^{2}}|{10: ^{2}}│\n"
                "│{11: ^{2}}|{12: ^{2}}│\n"
                "│{13: ^{2}}|{14: ^{2}}│\n"
                "│{15: ^{2}}|{16: ^{2}}│\n"
                "└{17:─^{1}}┘\n", "Launch info", 51, 25,
                "Blocks", blocks,
                "Block Size", blockSize,
                "Warps per Block", warpsPerBlock,
                "Buckets", buckets,
                "Bucket Size", bucketSize,
                "Buckets Memory (MB)", ((double) (binsMemory) / 1000000.0),
                "Level", (warpBased ? "Warp" : "Block"), ""
        );

        hipDeviceReset();

        unsigned int* deviceDiagonals;

        unsigned int* deviceOffsets;
        unsigned int* deviceSizes;
        unsigned int* deviceElements;
        unsigned int* deviceCounts;
        unsigned int* deviceBins;

        DeviceTimer deviceTimer;

        EventPair* devMemAlloc = deviceTimer.add("Device memory allocation");
        errorCheck(hipMalloc((void**)&deviceOffsets, sizeof(unsigned int) * d->cardinality))
        errorCheck(hipMalloc((void**)&deviceSizes, sizeof(unsigned int) * d->cardinality))
        errorCheck(hipMalloc((void**)&deviceElements, sizeof(unsigned int) * d->totalElements))
        errorCheck(hipMalloc((void**)&deviceCounts, sizeof(unsigned int) * combination(d->cardinality, 2)))
        errorCheck(hipMemset(deviceCounts, 0, sizeof(unsigned int) * combination(d->cardinality, 2)))
        errorCheck(hipMalloc((void**)&deviceBins, binsMemory))
        errorCheck(hipMemset(deviceBins, 0, binsMemory))
        if (partition) {
            errorCheck(hipMalloc((void**)&deviceDiagonals, sizeof(unsigned int) * 2 * (blocks + 1) * (combination(d->cardinality, 2))))
            errorCheck(hipMemset(deviceDiagonals, 0, sizeof(unsigned int) * 2 * (blocks + 1) * (combination(d->cardinality, 2))))
        }
        DeviceTimer::finish(devMemAlloc);

        EventPair* dataTransfer = deviceTimer.add("Transfer to device");
        errorCheck(hipMemcpy(deviceSizes, d->sizes, sizeof(unsigned int) * d->cardinality, hipMemcpyHostToDevice))
        errorCheck(hipMemcpy(deviceOffsets, deviceSizes, sizeof(unsigned int) * d->cardinality, hipMemcpyDeviceToDevice))
        errorCheck(hipMemcpy(deviceElements, d->elements, sizeof(unsigned int) * d->totalElements, hipMemcpyHostToDevice))
        DeviceTimer::finish(dataTransfer);

        EventPair* setOffsets = deviceTimer.add("Compute set offsets");
        thrust::exclusive_scan(thrust::device, deviceOffsets, deviceOffsets + d->cardinality, deviceOffsets, 0); // in-place scan
        DeviceTimer::finish(setOffsets);

        if (partition) {
            EventPair *findDiags = deviceTimer.add("Find diagonals");
            findDiagonals<<<blocks, 32>>>(d->cardinality, deviceElements, deviceSizes, deviceOffsets, deviceDiagonals, deviceCounts);
            DeviceTimer::finish(findDiags);
        }

        EventPair* hashInter = deviceTimer.add("Hash intersection");
        if (partition) {
            intersectPathHI<<<blocks, blockSize, sizeof(unsigned int) * buckets>>>
                (d->cardinality, deviceElements, deviceOffsets, deviceCounts, deviceBins, buckets, bucketSize, deviceDiagonals);
        } else {
            if (warpBased) {
                warpBasedHI<<<blocks, blockSize, sizeof(unsigned int) * buckets * warpsPerBlock>>>(d->cardinality, deviceElements, deviceSizes,
                                                                                                                deviceOffsets, deviceCounts, deviceBins, buckets, bucketSize);
            } else {
                blockBasedHI<false><<<blocks, blockSize, sizeof(unsigned int) * buckets>>>(d->cardinality, deviceElements, deviceSizes,
                                                                                                        deviceOffsets, deviceCounts, deviceBins, buckets, bucketSize);
            }
        }
        DeviceTimer::finish(hashInter);


        std::vector<unsigned int> counts(combination(d->cardinality, 2));

        EventPair* countTransfer = deviceTimer.add("Transfer result");
        errorCheck(hipMemcpy(&counts[0], deviceCounts, sizeof(unsigned int) * combination(d->cardinality, 2), hipMemcpyDeviceToHost))
        DeviceTimer::finish(countTransfer);

        EventPair* freeDevMem = deviceTimer.add("Free device memory");
        errorCheck(hipFree(deviceBins))
        errorCheck(hipFree(deviceOffsets))
        errorCheck(hipFree(deviceSizes))
        errorCheck(hipFree(deviceElements))
        errorCheck(hipFree(deviceCounts))
        if (partition) {
            errorCheck(hipFree(deviceDiagonals))
        }
        DeviceTimer::finish(freeDevMem);

        hipDeviceSynchronize();

        deviceTimer.print();

        if (!output.empty()) {
            fmt::print("Writing result to file {}\n", output);
            writeResult(d->cardinality, counts, output);
            fmt::print("Finished\n");
        }

    } catch (const cxxopts::OptionException& e) {
        fmt::print("{}\n", e.what());
        return 1;
    }
    return 0;
}

__global__ void warpBasedHI(unsigned int numOfSets, const unsigned int* elements, const unsigned int* sizes,
                            const unsigned int* offsets, unsigned int* counts, unsigned int* bins, unsigned int numOfBuckets, unsigned int bucketSize) {
    extern unsigned int __shared__ s[]; // [256*4]
    unsigned int* binCounts = s;

    unsigned int warpSize = 32;
    unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int warpsPerBlock = blockDim.x / warpSize;
    unsigned int globalWarpId = globalThreadId / warpSize;
    unsigned int warpId = threadIdx.x / warpSize; // local warp id
    unsigned int binSize = numOfBuckets * bucketSize;
    unsigned int binStart = globalWarpId * binSize;
    unsigned int binOffset = warpId * numOfBuckets;

    // Sets must be sorted in ascending order, thus set a will always be smaller than set b in order to minimize collisions
    // and avoid redundant work
    for (unsigned int a = globalWarpId; a < numOfSets - 1; a += gridDim.x * warpsPerBlock) {
        unsigned int aStart = offsets[a];
        unsigned int aEnd = offsets[a + 1];

        __syncwarp();

        // ensure bit counts are empty
        for (unsigned int i = threadIdx.x % warpSize + binOffset; i < binOffset + numOfBuckets; i += warpSize) {
            binCounts[i] = 0;
        }

        __syncwarp();

        // Hash shorter set
        for (unsigned int i = threadIdx.x % warpSize + aStart; i < aEnd; i += warpSize) {
            unsigned int element = elements[i];
            unsigned int bin = element % numOfBuckets;
            unsigned int index = atomicAdd(&binCounts[bin + binOffset], 1);
            bins[index * numOfBuckets + bin + binStart] = element;
        }

        __syncwarp();

        for (unsigned int b = a + 1; b < numOfSets; b++) {

            unsigned int bStart = offsets[b];
            unsigned int bEnd = bStart + sizes[b];

            unsigned int count = 0;

            // probe larger set
            for (unsigned int i = threadIdx.x % warpSize + bStart; i < bEnd; i += warpSize) {
                unsigned int element = elements[i];
                unsigned int bin = element % numOfBuckets;
                count += linearSearch(element, bins, binCounts, bin, binOffset, binStart, numOfBuckets);
            }

            if (count > 0) {
                atomicAdd(counts + triangular_idx(numOfSets, a, b), count);
            }

            __syncwarp();
        }

    }
}

template <bool split>
__global__ void blockBasedHI(unsigned int numOfSets, const unsigned int* elements, const unsigned int* sizes,
                             const unsigned int* offsets, unsigned int* counts, unsigned int* bins, unsigned int numOfBuckets, unsigned int bucketSize)
{
    extern unsigned int __shared__ s[];
    unsigned int* binCounts = s;

    unsigned int binSize = numOfBuckets * bucketSize;
    unsigned int binStart = blockIdx.x * binSize;

    // Sets must be sorted in ascending order, thus set a will always be smaller than set b in order to minimize collisions
    // and avoid redundant work
    for (unsigned int a = (split ? 0 : blockIdx.x); a < numOfSets - 1; a += (split ? 1 : gridDim.x)) {

        unsigned int aStart = offsets[a];
        unsigned int aEnd = offsets[a + 1];

        // ensure bit counts are empty
        for (unsigned int i = threadIdx.x; i < numOfBuckets; i += blockDim.x) {
            binCounts[i] = 0;
        }

        __syncthreads();

        // Hash shorter set
        for (unsigned int i = threadIdx.x + aStart; i < aEnd; i += blockDim.x) {
            unsigned int element = elements[i];
            unsigned int bin = element % numOfBuckets;
            unsigned int index = atomicAdd(&binCounts[bin], 1);
            bins[index * numOfBuckets + bin + binStart] = element;
        }

        __syncthreads();

        for (unsigned int b = (split ? a + blockIdx.x : a + 1); b < numOfSets; b += (split ? gridDim.x : 1)) {

            unsigned int bStart = offsets[b];
            unsigned int bEnd = bStart + sizes[b];

            unsigned int count = 0;

            // probe larger set
            for (unsigned int i = threadIdx.x + bStart; i < bEnd; i += blockDim.x) {
                unsigned int element = elements[i];
                unsigned int bin = element % numOfBuckets;
                count += linearSearch(element, bins, binCounts, bin, 0, binStart, numOfBuckets);
            }

            if (count > 0) {
                atomicAdd(counts + triangular_idx(numOfSets, a, b), count);
            }
            __syncthreads();
        }
    }

}


__global__ void intersectPathHI(unsigned int numOfSets, const unsigned int* elements, const unsigned int* offsets,
                                unsigned int* counts, unsigned int* bins, unsigned int numOfBuckets,
                                unsigned int bucketSize, unsigned int* globalDiagonals)
{
    extern unsigned int __shared__ s[];
    unsigned int* binCounts = s;

    unsigned int binSize = numOfBuckets * bucketSize;
    unsigned int binStart = blockIdx.x * binSize;

    for (unsigned int a = 0; a < numOfSets - 1; a++) {
        for (unsigned int b = a + 1; b < numOfSets; b++) { // iterate every combination
            unsigned int *diagonals = globalDiagonals + (2 * (gridDim.x + 1)) * triangular_idx(numOfSets, a, b);

            unsigned int aOffset = offsets[a];
            unsigned int bOffset = offsets[b];

            unsigned int aStart = diagonals[blockIdx.x];
            unsigned int aEnd = diagonals[blockIdx.x + 1];

            unsigned int bStart = diagonals[(gridDim.x + 1) + blockIdx.x];
            unsigned int bEnd = diagonals[(gridDim.x + 1) + blockIdx.x + 1];

            // ensure bit counts are empty
            for (unsigned int i = threadIdx.x; i < numOfBuckets; i += blockDim.x) {
                binCounts[i] = 0;
            }

            __syncthreads();

            // Hash shorter set
            for (unsigned int i = threadIdx.x + aStart; i < aEnd; i += blockDim.x) {
                unsigned int element = (elements + aOffset)[i];
                unsigned int bin = element % numOfBuckets;
                unsigned int index = atomicAdd(&binCounts[bin], 1);
                bins[index * numOfBuckets + bin + binStart] = element;
            }

            __syncthreads();

            unsigned int count = 0;

            // probe larger set
            for (unsigned int i = threadIdx.x + bStart; i < bEnd; i += blockDim.x) {
                unsigned int element = (elements + bOffset)[i];
                unsigned int bin = element % numOfBuckets;
                count += linearSearch(element, bins, binCounts, bin, 0, binStart, numOfBuckets);
            }

            if (count > 0) {
                atomicAdd(counts + triangular_idx(numOfSets, a, b), count);
            }
            __syncthreads();

        }
    }
}
